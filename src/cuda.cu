#include "hip/hip_runtime.h"
// cuda.cu
#include "hip/hip_runtime.h"
#include "utils.h"
#include <opencv2/opencv.hpp>

#include <chrono>
#include <iostream>

using namespace cv;
using namespace std;

void allocate_memory(short int*& pointer, int height, int width){
    hipMallocManaged(&pointer, height*width*sizeof(short int));
}

void clear_memory(short int*& pointer){
    hipFree(pointer);
}

void createGaussianKernel(float*& kernel , float sigma, int window){
    int center = (window)/2;
    float x;
    float sum = 0.0;

    for(int i = 0; i < window; i++){
        x = float(i - center);
        float product = exp(-((x*x)/(2*sigma*sigma)))/(sqrt(6.2831853)*sigma);
        kernel[i] = product;
        sum += product;
    }

    for(int i = 0; i < window; i++){
        kernel[i] /= sum;
    }
}

__global__ void gaussian_util(unsigned char* img, float sigma, int window, int height, int width, float* kernel, float* temp_img, short int* result){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;
    
    int center = window/2;

    // Blur in the x direction
    for(int row = idx_y; row < height; row += stride_y){
        for(int col = idx_x; col < width; col += stride_x){
            int idx = row * width + col;
            float sum = 0;
            float count = 0;
            for(int k = -center; k < (center + 1); k++){
                if(((col+k) >= 0) and (col+k) < width){
                    sum += ((float)(img[idx+k]) * kernel[center+k]);
                    count += kernel[center+k];
                }
            }
            temp_img[idx] = (sum/count);
        }
    }
    __syncthreads();

    // Blur in the y direction
    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){
            int idx = row * width + col;
            float sum = 0;
            float count = 0;
            for(int k = -center; k < (center + 1); k++){
                if(((row + k) >= 0) and ((row + k) < height)){
                    sum += ((float)(temp_img[idx + (k*width)]) * kernel[center+k]);
                    count += kernel[center+k];
                }
            }
            result[idx] = (short int)(sum/count);
        }
    }
}

void cuda_gaussian(unsigned char* img, float sigma, int rows, int columns, short int*& result){
    unsigned char* shared_img;
    float* temp_img; 
    int window = 1 + 2 * ceil(3 * sigma);
    float* kernel;
    
    hipMallocManaged(&shared_img, rows*columns*sizeof(unsigned char));
    hipMallocManaged(&temp_img, rows*columns*sizeof(float));
    hipMallocManaged(&result, rows*columns*sizeof(short int));
    hipMallocManaged(&kernel, window * sizeof(float));

    createGaussianKernel(kernel, sigma, window);

    for(int i = 0; i < rows*columns; i++){
        shared_img[i] = img[i];
    }

    gaussian_util<<<3,512>>>(shared_img, sigma, window, rows, columns, kernel, temp_img, result);

    hipDeviceSynchronize();

    hipFree(shared_img);
    hipFree(temp_img);
    hipFree(kernel);
}

__global__ void xy_utility(short int* img, int height, int width, short int* grad_x, short int* grad_y){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;
    
    // Gradient in the x direction, filter in the form of:
    // -1   0   1
    // -2   0   2
    // -1   0   1
    for(int row = idx_y; row < height; row += stride_y){
        for(int col = idx_x; col < width; col += stride_x){
            int pos = row * width + col;
            if(col == 0){
                // Leftmost column, all rows; pad out of bounds with border values
                grad_x[pos] = (2 * img[pos+1]) - (2 * img[pos]);
                // include row above
                if(row != height-1){
                    grad_x[pos] += (img[pos+width+1] - img[pos+width]);
                }
                // include row below
                if(row != 0){
                    grad_x[pos] += (img[pos-width+1] - img[pos-width]);
                }
            }
            else if(col == (width-1)){
                // Rightmost column, all rows; pad out of bounds with border values
                grad_x[pos] = (2 * img[pos]) - (2 * img[pos-1]);
                // include row below
                if(row != height-1){
                    grad_x[pos] += (img[pos+width] - img[pos+width-1]);
                }
                // include row above
                if(row != 0){
                    grad_x[pos] += (img[pos-width] - img[pos-width-1]);
                }
            }
            else{
                grad_x[pos] = (2 * img[pos+1]) - (2 * img[pos-1]);
                if(row != height-1){
                    grad_x[pos] += (img[pos+width+1] - img[pos+width-1]);
                }
                if(row != 0){
                    grad_x[pos] += (img[pos-width+1] - img[pos-width-1]);
                }
            }
        }
    }

    // Gradient in the y direction, filter in the form of:
    //  1   2   1
    //  0   0   0
    // -1  -2  -1
    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){
            int pos = row * width + col;
            if(row == 0){
                // Topmost row, all columns; pad out of bounds with border values
                grad_y[pos] = (2 * img[pos+width]) - (2 * img[pos]);
                if(col != width-1){
                    grad_y[pos] += (img[pos+width+1]-img[pos+1]);
                }
                if(col != 0){
                    grad_y[pos] += (img[pos+width-1]-img[pos-1]);
                }
            }
            else if(row == (height - 1)){
                // Bottommost row, all columns; pad out of bounds with border values
                grad_y[pos] = (2*img[pos]) - (2*img[pos-width]);
                if(col != width-1){
                    grad_y[pos] += (img[pos+1] - img[pos-width+1]);
                }
                if(col != 0){
                    grad_y[pos] += (img[pos-1]-img[pos-width-1]);
                }
            }
            else{
                // Middle, nonborder pixels
                grad_y[pos] = (2*img[pos+width]) - (2*img[pos-width]);
                if(col != width-1){
                    grad_y[pos] += (img[pos+width+1]-img[pos-width+1]);
                }
                if(col != 0){
                    grad_y[pos] += (img[pos+width-1]-img[pos-width-1]);
                }
            }
        }
    }
}

void cuda_calculate_xy_gradient(short int* img, int height, int width, short int*& grad_x, short int*& grad_y){
    hipMallocManaged(&grad_x, height*width*sizeof(short int));
    hipMallocManaged(&grad_y, height*width*sizeof(short int));

    xy_utility<<<3,512>>>(img, height, width, grad_x, grad_y);
    
    hipDeviceSynchronize();

    hipFree(img);
}

__global__ void sobel_utility(short int* grad_x, short int* grad_y, int height, int width, short int* magnitude, short int* angle){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){
            // Calculate magnitude of gradient at every pixel
            int idx = row * width + col;
            magnitude[idx] = (int)sqrtf((grad_x[idx] * grad_x[idx]) + (grad_y[idx] * grad_y[idx]));

            // Calculate angle of gradient at every pixel
            float temp_angle = atan2((double)grad_y[idx],(double)grad_x[idx]);
            temp_angle *= (180/PI);
            if(temp_angle < 0){
                temp_angle = 360 + temp_angle;
            }
            if((temp_angle >= 22.5 && temp_angle < 67.5) || (temp_angle >= 202.5 && temp_angle < 247.5)){
                angle[idx] = 45;
            }
            else if((temp_angle >= 112.5 && temp_angle < 157.5) || (temp_angle >= 292.5 && temp_angle < 337.5)){
                angle[idx] = 135;
            }
            else if((temp_angle >= 67.5 && temp_angle < 112.5) || (temp_angle >= 247.5 && temp_angle < 292.5)){
                angle[idx] = 90;
            }
            else{
                angle[idx] = 0;
            }
        }
    }
}

void cuda_sobel_operator(short int* grad_x, short int* grad_y, int height, int width, short int*& magnitude, short int*& angle){
    hipMallocManaged(&magnitude, height*width*sizeof(short int));
    hipMallocManaged(&angle, height*width*sizeof(short int));

    sobel_utility<<<3,512>>>(grad_x, grad_y, height, width, magnitude, angle);

    hipDeviceSynchronize();

    hipFree(grad_x);
    hipFree(grad_y);
}

__global__ void nonmaximal_utility(short int* magnitude, short int* angle, int height, int width, short int* result){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){

            int idx = row * width + col;
            bool max = true;

            if(angle[idx] == 0){
                int left = idx - 1;
                int right = idx + 1;
    
                if((idx%width) > 0){
                    if(magnitude[idx] <= magnitude[left]){max = false;}
                }
                if(idx%width < width-1){
                    if(magnitude[idx] <= magnitude[right]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            }   
            else if(angle[idx] == 45){
                int upRight = idx + 1 - width;
                int downLeft = idx - 1 + width;
    
                if((idx%width < width-1) && (idx >= height)){
                    if(magnitude[idx] <= magnitude[upRight]){max = false;}
                }
                if((idx%width > 0) && (idx < (height*width)-height)){
                    if(magnitude[idx] <= magnitude[downLeft]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            }
            else if(angle[idx] == 90){
                int up = idx - width;
                int down = idx + width;
    
                if(idx >= height){
                    if(magnitude[idx] <= magnitude[up]){max = false;}
                }
                if(idx < (height*width)-height){
                    if(magnitude[idx] <= magnitude[down]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            } 
            else if(angle[idx] == 135){
                int upLeft = idx - 1 - width;
                int downRight = idx + 1 + width;
    
                if((idx%width > 0) && (idx >= height)){
                    if(magnitude[idx] <= magnitude[upLeft]){max = false;}
                }
                if((idx%width < width-1) && (idx < (height*width)-height)){
                    if(magnitude[idx] <= magnitude[downRight]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            }
        }
    }
}

void cuda_nonmaixmal_suppression(short int* magnitude, short int* angle, int height, int width, short int*& result){
    hipMallocManaged(&result, height*width*sizeof(short int));

    nonmaximal_utility<<<3,512>>>(magnitude,angle,height,width,result);

    hipDeviceSynchronize();

    hipFree(magnitude);
    hipFree(angle);
}

void cuda_canny(unsigned char* img, float sigma, int minVal, int maxVal, int height, int width){
    short int* smoothed_img;    // Image blurred by a Gaussian filter
    short int* grad_x;
    short int* grad_y;
    short int* magnitude;       // Magnitude of edges, calculated as sqrt(grad_x^2 + grad_y^2)
    short int* angle;           // Angle/direction of edges, calculated as arctan2(grad_y, grad_x)
    short int* nonmaximal;      // Edges w/ nonmaximal suppression applied to neighbors in angle direction

    cuda_gaussian(img,sigma,height,height,smoothed_img);

    if(STEPS){
        Mat gaussianMat(256,256, CV_16S, smoothed_img);
        Mat gaussian_display;

        normalize(gaussianMat, gaussian_display, 0, 255, NORM_MINMAX);
        gaussian_display.convertTo(gaussian_display, CV_8U);

        imshow("CudaGaussian Visual Test", gaussian_display);
        waitKey(0);
    }

    cuda_calculate_xy_gradient(smoothed_img, height, height, grad_x, grad_y);

    if(STEPS){
        Mat xMat(256,256, CV_16S, grad_x);
        Mat yMat(256,256, CV_16S, grad_y);
        Mat x_display, y_display;

        normalize(xMat, x_display, 0, 255, NORM_MINMAX);
        x_display.convertTo(x_display, CV_8U);

        normalize(yMat, y_display, 0, 255, NORM_MINMAX);
        y_display.convertTo(y_display, CV_8U);

        imshow("X Gradient Visual Test", x_display);
        waitKey(0);

        imshow("Y Gradient Visual Test", y_display);
        waitKey(0);

    }
    
    cuda_sobel_operator(grad_x, grad_y, height, height, magnitude, angle);
    cuda_nonmaixmal_suppression(magnitude, angle, height, height, nonmaximal);

    if(STEPS){
        Mat result_mat(256,256, CV_16S, nonmaximal);
        Mat result_display;

        normalize(result_mat, result_display, 0, 255, NORM_MINMAX);
        result_display.convertTo(result_display, CV_8U);

        imshow("Nonmaximal Visual Test", result_display);
        waitKey(0);
    }

    hysteresis(nonmaximal, height, width, minVal, maxVal);

    // Display final image with canny edge detection applied to it
    Mat finalMat(height,width, CV_16S, nonmaximal);
    Mat final_display;
    normalize(finalMat, final_display, 0, 255, NORM_MINMAX);
    final_display.convertTo(final_display, CV_8U);
    imshow("Nonmaximal Image", final_display);
    waitKey(0);

    clear_memory(nonmaximal);
}
    