// cuda.cu
#include "hip/hip_runtime.h"
#include <iostream>

void clear_memory(short int*& pointer){
    hipFree(pointer);
}

void createGaussianKernel(float*& kernel , float sigma, int window){
    int center = (window)/2;
    float x;
    float sum = 0.0;

    for(int i = 0; i < window; i++){
        x = float(i - center);
        float product = exp(-((x*x)/(2*sigma*sigma)))/(sqrt(6.2831853)*sigma);
        kernel[i] = product;
        sum += product;
    }

    for(int i = 0; i < window; i++){
        kernel[i] /= sum;
    }
}

__global__ void gaussian_util(unsigned char* img, float sigma, int window, int rows, int columns, float*& kernel, float* temp_img, short int* result){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = idx_y * columns + idx_x;
    int stride = blockDim.x * gridDim.x;
    
    int center = window/2;

    // Blur in the x direction
    for(int i = idx; i < rows * columns; i+= stride){
        float sum = 0;
        float count = 0;
        for(int k = -center; k < (center + 1); k++){
            if(((i+k)%columns >= 0) and (i+k)%columns < columns){
                sum += (float(img[idx + k]) * kernel[center + k]);
                count += kernel[center+k];
            }
        }
        temp_img[idx] = sum/count;
    }
    __syncthreads();

    // Blur in the y direction
    for(int i = idx; i < rows * columns; i += stride){
        float sum = 0;
        float count = 0;
        for(int k = -center; k < (center + 1); k++){
            if(((i+(k*columns)) >= columns) and ((i+(k*columns)) < (rows*columns)-columns)){
                sum += (float(temp_img[i+(k*columns)]) * kernel[center+k]);
                count += kernel[center+k];
            }
        }
        result[idx] = (short int)(sum/count);
    }

    __syncthreads();
}

void gaussian(unsigned char* img, float sigma, int rows, int columns, short int*& result){
    unsigned char* shared_img;
    float* temp_img; 
    int window = 1 + 2 * ceil(3 * sigma);
    float* kernel;
    
    hipMallocManaged(&shared_img, rows*columns*sizeof(unsigned char));
    hipMallocManaged(&temp_img, rows*columns*sizeof(float));
    hipMallocManaged(&result, rows*columns*sizeof(short int));
    hipMallocManaged(&kernel, window * sizeof(float));

    createGaussianKernel(kernel, sigma, window);

    for(int i = 0; i < rows*columns; i++){
        shared_img[i] = img[i];
    }
    std::cout << kernel[0] << " " << kernel[1] << std::endl;

    gaussian_util<<<1,1>>>(shared_img, sigma, window, rows, columns, kernel, temp_img, result);

    std::cout << result[0] << std::endl;

    hipFree(shared_img);
    hipFree(temp_img);
    hipFree(kernel);
}



