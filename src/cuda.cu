#include "hip/hip_runtime.h"
// cuda.cu
#include "hip/hip_runtime.h"
#include "utils.h"
#include <opencv2/opencv.hpp>

#include <chrono>
#include <iostream>

#define NUM_BLOCKS 20
#define BLOCK_SIZE 32

using namespace cv;
using namespace std;

void createGaussianKernel(float*& kernel , float sigma, int window){
    int center = (window)/2;
    float x;
    float sum = 0.0;

    for(int i = 0; i < window; i++){
        x = float(i - center);
        float product = exp(-((x*x)/(2*sigma*sigma)))/(sqrt(6.2831853)*sigma);
        kernel[i] = product;
        sum += product;
    }

    for(int i = 0; i < window; i++){
        kernel[i] /= sum;
    }
}

__global__ void gaussian_util(unsigned char* img, float sigma, int window, int height, int width, float* kernel, float* temp_img, short int* result){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;
    
    int center = window/2;

    // Blur in the x direction
    for(int row = idx_y; row < height; row += stride_y){
        for(int col = idx_x; col < width; col += stride_x){
            int idx = row * width + col;
            float sum = 0;
            float count = 0;
            for(int k = -center; k < (center + 1); k++){
                if(((col+k) >= 0) and (col+k) < width){
                    sum += ((float)(img[idx+k]) * kernel[center+k]);
                    count += kernel[center+k];
                }
            }
            temp_img[idx] = (sum/count);
        }
    }
    __syncthreads();

    // Blur in the y direction
    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){
            int idx = row * width + col;
            float sum = 0;
            float count = 0;
            for(int k = -center; k < (center + 1); k++){
                if(((row + k) >= 0) and ((row + k) < height)){
                    sum += ((float)(temp_img[idx + (k*width)]) * kernel[center+k]);
                    count += kernel[center+k];
                }
            }
            result[idx] = (short int)(sum/count);
        }
    }
}

void cuda_gaussian(unsigned char*& img, float sigma, int rows, int columns, short int*& result_host){
    unsigned char* img_device;
    float* temp_device; 
    int window = 1 + 2 * ceil(3 * sigma);
    float* kernel;
    short int* result_device;
    result_host = new short int[rows*columns];
    
    hipMalloc(&img_device, rows*columns*sizeof(unsigned char));
    hipMalloc(&temp_device, rows*columns*sizeof(float));
    hipMalloc(&result_device, rows*columns*sizeof(short int));
    hipMallocManaged(&kernel, window * sizeof(float));

    createGaussianKernel(kernel, sigma, window);

    hipMemcpy(img_device, img, rows*columns*sizeof(unsigned char), hipMemcpyHostToDevice);

    gaussian_util<<<NUM_BLOCKS,BLOCK_SIZE>>>(img_device, sigma, window, rows, columns, kernel, temp_device, result_device);

    hipDeviceSynchronize();

    hipMemcpy(result_host, result_device, rows*columns*sizeof(short int), hipMemcpyDeviceToHost);

    hipFree(img_device);
    hipFree(temp_device);
    hipFree(kernel);
    hipFree(result_device);
}

__global__ void sobel_util(short int* img, int height, int width, short int* magnitude, short int* angle){
    int blk_x = blockIdx.x;
    int blk_y = blockIdx.y;

    int thrd_x = threadIdx.x;
    int thrd_y = threadIdx.y;

    // int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    // int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = gridDim.x;
    int stride_y = gridDim.y;


    __shared__ short int img_shared[BLOCK_SIZE+2][BLOCK_SIZE+2];

    __shared__ short int grad_x[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ short int grad_y[BLOCK_SIZE][BLOCK_SIZE];
    
    for(int row = blk_y; row < height/BLOCK_SIZE; row += stride_y){
        for(int col = blk_x; col < width/BLOCK_SIZE; col += stride_x){
            // Position relative to entire image
            int img_pos = (((row * BLOCK_SIZE * width)+(thrd_y * width)) + ((col * BLOCK_SIZE) + thrd_x));

            img_shared[thrd_y+1][thrd_x+1] = img[img_pos];

            // Fill in outer border
            // left/right border
            if(blk_x == 0){
                img_shared[thrd_y+1][thrd_x] = img[img_pos];
            }else{
                img_shared[thrd_y+1][thrd_x] = img[img_pos - 1];
            }
            if(blk_x == ((width/BLOCK_SIZE)-1)){
                img_shared[thrd_y+1][thrd_x+2] = -1;
            }else{
                img_shared[thrd_y+1][thrd_x+2] = img[img_pos+1];
            }
            
            // top/bottom border
            if(blk_y == 0){
                img_shared[thrd_y][thrd_x+1] = img[img_pos];
            }else{
                img_shared[thrd_y][thrd_x+1] = img[img_pos - width];
            }
            if(blk_y == ((height/BLOCK_SIZE)-1)){
                img_shared[thrd_y+2][thrd_x+1] = -1;
            }else{
                img_shared[thrd_y+2][thrd_x+1] = img[img_pos + width];
            }
            
            // corners
            if(blk_x == 0 and blk_y == 0){
                img_shared[thrd_y][thrd_x] = img[img_pos];
            }else{
                img_shared[thrd_y][thrd_x] = img[img_pos - 1 - width];
            }

            if(blk_x == 0 and blk_y == ((height/BLOCK_SIZE)-1)){
                img_shared[thrd_y+2][thrd_x] = img[img_pos];
            }else{
                img_shared[thrd_y+2][thrd_x] = img[img_pos -1 + width];
            }

            if(blk_x == ((width/BLOCK_SIZE)-1) and blk_y == 0){
                img_shared[thrd_y][thrd_x+2] = img[img_pos];
            }else{
                img_shared[thrd_y][thrd_x+2] = img[img_pos +1 - width];
            }

            if(blk_x == ((width/BLOCK_SIZE)-1) and blk_y == ((height/BLOCK_SIZE)-1)){
                img_shared[thrd_y+2][thrd_x+2] = img[img_pos];
            }else{
                img_shared[thrd_y+2][thrd_x+2] = img[img_pos +1 + width];
            }

            __syncthreads();

            grad_x[thrd_y][thrd_x] = (2 * img_shared[thrd_y+1][thrd_x+2]) - (2 * img_shared[thrd_y+1][thrd_x]);
            grad_x[thrd_y][thrd_x] += img_shared[thrd_y+2][thrd_x+2] - img_shared[thrd_y+2][thrd_x];
            grad_x[thrd_y][thrd_x] += img_shared[thrd_y][thrd_x+2] - img_shared[thrd_y][thrd_x];

            grad_y[thrd_y][thrd_x] = (2 * img_shared[thrd_y][thrd_x+1]) - (2 * img_shared[thrd_y+2][thrd_x+1]);
            grad_y[thrd_y][thrd_x] += img_shared[thrd_y][thrd_x+2] - img_shared[thrd_y+2][thrd_x+2];
            grad_y[thrd_y][thrd_x] += img_shared[thrd_y][thrd_x] - img_shared[thrd_y+2][thrd_x];

            magnitude[img_pos] = (short int)sqrtf((grad_x[thrd_y][thrd_x] * grad_x[thrd_y][thrd_x]) 
                                                + (grad_y[thrd_y][thrd_x] * grad_y[thrd_y][thrd_x]));

            // Calculate angle of gradient at every pixel
            float temp_angle = atan2((double)grad_y[thrd_y][thrd_x],(double)grad_x[thrd_y][thrd_x]);
            temp_angle *= (180/PI);
            if(temp_angle < 0){
                temp_angle = 360 + temp_angle;
            }
            if((temp_angle >= 22.5 && temp_angle < 67.5) || (temp_angle >= 202.5 && temp_angle < 247.5)){
                angle[img_pos] = 45;
            }
            else if((temp_angle >= 112.5 && temp_angle < 157.5) || (temp_angle >= 292.5 && temp_angle < 337.5)){
                angle[img_pos] = 135;
            }
            else if((temp_angle >= 67.5 && temp_angle < 112.5) || (temp_angle >= 247.5 && temp_angle < 292.5)){
                angle[img_pos] = 90;
            }
            else{
                angle[img_pos] = 0;
            }
        }
    }
}

void cuda_sobel(short int*& img_host, int height, int width, short int*& magnitude_host, short int*& angle_host){
    short int* img_device;
    short int* grad_x_device;
    short int* grad_y_device;
    short int* magnitude_device;
    short int* angle_device;

    magnitude_host = new short int[height*width];
    angle_host = new short int[height*width];

    hipMalloc(&img_device, height*width*sizeof(short int));
    hipMemcpy(img_device, img_host, height*width*sizeof(short int), hipMemcpyHostToDevice);

    hipMalloc(&grad_x_device, height*width*sizeof(short int));
    hipMalloc(&grad_y_device, height*width*sizeof(short int));
    hipMalloc(&magnitude_device, height*width*sizeof(short int));
    hipMalloc(&angle_device, height*width*sizeof(short int));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(width/dimBlock.x, height/dimBlock.y);
    sobel_util<<<dimGrid,dimBlock>>>(img_device, height, width, magnitude_device, angle_device);
    
    hipDeviceSynchronize();

    hipMemcpy(magnitude_host, magnitude_device, height*width*sizeof(short int), hipMemcpyDeviceToHost);
    hipMemcpy(angle_host, angle_device, height*width*sizeof(short int), hipMemcpyDeviceToHost);

    hipFree(angle_device);
    hipFree(magnitude_device);
    hipFree(grad_x_device);
    hipFree(grad_y_device);
    hipFree(img_device);
    delete[] img_host;
}


__global__ void nonmaximal_utility(short int* magnitude, short int* angle, int height, int width, short int* result){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){

            int idx = row * width + col;
            bool max = true;

            if(angle[idx] == 0){
                int left = idx - 1;
                int right = idx + 1;
    
                if((idx%width) > 0){
                    if(magnitude[idx] <= magnitude[left]){max = false;}
                }
                if(idx%width < width-1){
                    if(magnitude[idx] <= magnitude[right]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            }   
            else if(angle[idx] == 45){
                int upRight = idx + 1 - width;
                int downLeft = idx - 1 + width;
    
                if((idx%width < width-1) && (idx - width >= 0)){
                    if(magnitude[idx] <= magnitude[upRight]){max = false;}
                }
                if((idx%width > 0) && (idx + width < (height*width))){
                    if(magnitude[idx] <= magnitude[downLeft]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            }
            else if(angle[idx] == 90){
                int up = idx - width;
                int down = idx + width;
    
                if(idx - width >= 0){
                    if(magnitude[idx] <= magnitude[up]){max = false;}
                }
                if(idx + width < (height*width)){
                    if(magnitude[idx] <= magnitude[down]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            } 
            else if(angle[idx] == 135){
                int upLeft = idx - 1 - width;
                int downRight = idx + 1 + width;
    
                if((idx%width > 0) && (idx - width >= 0)){
                    if(magnitude[idx] <= magnitude[upLeft]){max = false;}
                }
                if((idx%width < width-1) && (idx + width < (height*width))){
                    if(magnitude[idx] <= magnitude[downRight]){max = false;}
                }
                if(max){result[idx] = magnitude[idx];}
                else{result[idx] = NOEDGE;}
            }
        }
    }
}

void cuda_nonmaixmal_suppression(short int*& magnitude_host, short int*& angle_host, int height, int width, short int*& result_host){
    short int* magnitude_device;
    short int* angle_device;
    short int* result_device;
    result_host = new short int[height*width];

    hipMalloc(&magnitude_device, height*width*sizeof(short int));
    hipMalloc(&angle_device, height*width*sizeof(short int));
    hipMalloc(&result_device, height*width*sizeof(short int));

    hipMemcpy(magnitude_device, magnitude_host, height*width*sizeof(short int), hipMemcpyHostToDevice);
    hipMemcpy(angle_device, angle_host, height*width*sizeof(short int), hipMemcpyHostToDevice);

    nonmaximal_utility<<<NUM_BLOCKS,BLOCK_SIZE>>>(magnitude_device,angle_device,height,width,result_device);

    hipDeviceSynchronize();

    hipMemcpy(result_host, result_device, height*width*sizeof(short int), hipMemcpyDeviceToHost);

    hipFree(result_device);
    hipFree(magnitude_device);
    hipFree(angle_device);
    delete[] magnitude_host;
    delete[] angle_host;
}

void cuda_canny(unsigned char* img, float sigma, int minVal, int maxVal, int height, int width, bool steps){
    short int* smoothed_img;    // Image blurred by a Gaussian filter
    short int* magnitude;       // Magnitude of edges, calculated as sqrt(grad_x^2 + grad_y^2)
    short int* angle;           // Angle/direction of edges, calculated as arctan2(grad_y, grad_x)
    short int* nonmaximal;      // Edges w/ nonmaximal suppression applied to neighbors in angle direction

    cuda_gaussian(img,sigma,height,width,smoothed_img);

    if(steps){
        Mat gaussianMat(height,width, CV_16S, smoothed_img);
        Mat gaussian_display;

        normalize(gaussianMat, gaussian_display, 0, 255, NORM_MINMAX);
        gaussian_display.convertTo(gaussian_display, CV_8U);

        imshow("CudaGaussian Visual Test", gaussian_display);
        waitKey(0);
    }

    cuda_sobel(smoothed_img, height, width, magnitude, angle);

    if(steps){
        Mat sobel_mat(height,width, CV_16S, magnitude);
        Mat sobel_display;

        normalize(sobel_mat, sobel_display, 0, 255, NORM_MINMAX);
        sobel_display.convertTo(sobel_display, CV_8U);

        imshow("Sobel Visual Test", sobel_display);
        waitKey(0);
    }
    cuda_nonmaixmal_suppression(magnitude, angle, height, width, nonmaximal);

    if(steps){
        Mat result_mat(height,width, CV_16S, nonmaximal);
        Mat result_display;

        normalize(result_mat, result_display, 0, 255, NORM_MINMAX);
        result_display.convertTo(result_display, CV_8U);

        imshow("Nonmaximal Visual Test", result_display);
        waitKey(0);
    }

    hysteresis(nonmaximal, height, width, minVal, maxVal);

    // Display final image with canny edge detection applied to it
    Mat finalMat(height,width, CV_16S, nonmaximal);
    Mat final_display;
    normalize(finalMat, final_display, 0, 255, NORM_MINMAX);
    final_display.convertTo(final_display, CV_8U);
    imshow("Final Image", final_display);
    waitKey(0);

    delete[] nonmaximal;
}
    