// cuda.cu
#include "hip/hip_runtime.h"
#include <iostream>

void allocate_memory(short int*& pointer, int height, int width){
    hipMallocManaged(&pointer, height*width*sizeof(short int));
}

void clear_memory(short int*& pointer){
    hipFree(pointer);
}

void createGaussianKernel(float*& kernel , float sigma, int window){
    int center = (window)/2;
    float x;
    float sum = 0.0;

    for(int i = 0; i < window; i++){
        x = float(i - center);
        float product = exp(-((x*x)/(2*sigma*sigma)))/(sqrt(6.2831853)*sigma);
        kernel[i] = product;
        sum += product;
    }

    for(int i = 0; i < window; i++){
        kernel[i] /= sum;
    }
}

__global__ void gaussian_util(unsigned char* img, float sigma, int window, int height, int width, float* kernel, float* temp_img, short int* result){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;
    
    int center = window/2;

    // Blur in the x direction
    for(int row = idx_y; row < height; row += stride_y){
        for(int col = idx_x; col < width; col += stride_x){
            int idx = row * width + col;
            float sum = 0;
            float count = 0;
            for(int k = -center; k < (center + 1); k++){
                if(((col+k) >= 0) and (col+k) < width){
                    sum += ((float)(img[idx+k]) * kernel[center+k]);
                    count += kernel[center+k];
                }
            }
            temp_img[idx] = (sum/count);
        }
    }
    __syncthreads();

    // Blur in the y direction
    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){
            int idx = row * width + col;
            float sum = 0;
            float count = 0;
            for(int k = -center; k < (center + 1); k++){
                if(((row + k) >= 0) and ((row + k) < height)){
                    sum += ((float)(temp_img[idx + (k*width)]) * kernel[center+k]);
                    count += kernel[center+k];
                }
            }
            result[idx] = (short int)(sum/count);
        }
    }
}

void gaussian(unsigned char* img, float sigma, int rows, int columns, short int*& result){
    unsigned char* shared_img;
    float* temp_img; 
    int window = 1 + 2 * ceil(3 * sigma);
    float* kernel;
    
    hipMallocManaged(&shared_img, rows*columns*sizeof(unsigned char));
    hipMallocManaged(&temp_img, rows*columns*sizeof(float));
    hipMallocManaged(&result, rows*columns*sizeof(short int));
    hipMallocManaged(&kernel, window * sizeof(float));

    createGaussianKernel(kernel, sigma, window);

    for(int i = 0; i < rows*columns; i++){
        shared_img[i] = img[i];
    }

    gaussian_util<<<3,512>>>(shared_img, sigma, window, rows, columns, kernel, temp_img, result);

    hipDeviceSynchronize();

    hipFree(shared_img);
    hipFree(temp_img);
    hipFree(kernel);
}

__global__ void xy_utility(short int* img, int height, int width, short int* grad_x, short int* grad_y){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;
    
    // Gradient in the x direction, filter in the form of:
    // -1   0   1
    // -2   0   2
    // -1   0   1
    for(int row = idx_y; row < height; row += stride_y){
        for(int col = idx_x; col < width; col += stride_x){
            int pos = row * width + col;
            if(col == 0){
                // Leftmost column, all rows; pad out of bounds with border values
                grad_x[pos] = (2 * img[pos+1]) - (2 * img[pos]);
                // include row above
                if(row != height-1){
                    grad_x[pos] += (img[pos+width+1] - img[pos+width]);
                }
                // include row below
                if(row != 0){
                    grad_x[pos] += (img[pos-width+1] - img[pos-width]);
                }
            }
            else if(col == (width-1)){
                // Rightmost column, all rows; pad out of bounds with border values
                grad_x[pos] = (2 * img[pos]) - (2 * img[pos-1]);
                // include row below
                if(row != height-1){
                    grad_x[pos] += (img[pos+width] - img[pos+width-1]);
                }
                // include row above
                if(row != 0){
                    grad_x[pos] += (img[pos-width] - img[pos-width-1]);
                }
            }
            else{
                grad_x[pos] = (2 * img[pos+1]) - (2 * img[pos-1]);
                if(row != height-1){
                    grad_x[pos] += (img[pos+width+1] - img[pos+width-1]);
                }
                if(row != 0){
                    grad_x[pos] += (img[pos-width+1] - img[pos-width-1]);
                }
            }
        }
    }

    // Gradient in the y direction, filter in the form of:
    //  1   2   1
    //  0   0   0
    // -1  -2  -1
    for(int col = idx_x; col < width; col += stride_x){
        for(int row = idx_y; row < height; row += stride_y){
            int pos = row * width + col;
            if(row == 0){
                // Topmost row, all columns; pad out of bounds with border values
                grad_y[pos] = (2 * img[pos+width]) - (2 * img[pos]);
                if(col != width-1){
                    grad_y[pos] += (img[pos+width+1]-img[pos+1]);
                }
                if(col != 0){
                    grad_y[pos] += (img[pos+width-1]-img[pos-1]);
                }
            }
            else if(row == (height - 1)){
                // Bottommost row, all columns; pad out of bounds with border values
                grad_y[pos] = (2*img[pos]) - (2*img[pos-width]);
                if(col != width-1){
                    grad_y[pos] += (img[pos+1] - img[pos-width+1]);
                }
                if(col != 0){
                    grad_y[pos] += (img[pos-1]-img[pos-width-1]);
                }
            }
            else{
                // Middle, nonborder pixels
                grad_y[pos] = (2*img[pos+width]) - (2*img[pos-width]);
                if(col != width-1){
                    grad_y[pos] += (img[pos+width+1]-img[pos-width+1]);
                }
                if(col != 0){
                    grad_y[pos] += (img[pos+width-1]-img[pos-width-1]);
                }
            }
        }
    }
}

void cuda_calculate_xy_gradient(short int* img, int height, int width, short int*& grad_x, short int*& grad_y){
    hipMallocManaged(&grad_x, height*width*sizeof(short int));
    hipMallocManaged(&grad_y, height*width*sizeof(short int));

    xy_utility<<<3,512>>>(img, height, width, grad_x, grad_y);
    
    hipDeviceSynchronize();
    
    hipFree(img);
}


